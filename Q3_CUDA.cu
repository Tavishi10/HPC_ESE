#include "hip/hip_runtime.h"
//Find Factorial of a number using CUDA.
%%cu
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void calc_factorial(int *a, long int *fact)
{     
    int i;
    *fact=1;
    for(i=1;i<=*a;i++)
      *fact=*fact * i;
}

int main()
{
  int n=10;
  int *num;
  long int *res, result;  
  hipMalloc((void**)&num,sizeof(int));
  hipMalloc((void**)&res,sizeof(long int));
  hipMemcpy(num,&n,sizeof(int),hipMemcpyHostToDevice);
  calc_factorial<<<1,3>>>(num,res);
  hipMemcpy(&result,res,sizeof(long int),hipMemcpyDeviceToHost);
  printf("\nFactorial of number %d is %ld \n",n,result); 
  hipFree(num);     
  hipFree(res);
  return 0; 
}
